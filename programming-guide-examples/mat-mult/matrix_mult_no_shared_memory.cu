#include "hip/hip_runtime.h"
//Matrices are stored in row-major order:
//M (row, col) = *(M.elements + row*M.width + col)

#define BLOCK_SIZE 16

#define KERNEL_DEBUG 0

#include "main_func.h"

//Forward declaration of kernel! (prototyping)
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

void MatMul(const Matrix A, const Matrix B, Matrix C)
{
	//Load A, B to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(specified_precision);
	hipMalloc(&d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	size = B.width * B.height * sizeof(specified_precision);
	hipMalloc(&d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

	//Allocate C in device memory
	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	size = C.width * C.height * sizeof(specified_precision);
	hipMalloc(&d_C.elements, size);

	//Invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width/dimBlock.x, A.height / dimBlock.y);
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);


#if (KERNEL_DEBUG ==1)
	hipError_t hipError_t;
	hipError_t = hipGetLastError();
	if(hipError_t != hipSuccess)
	{
		printf("  hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
	}
#endif
	//Read C to host
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

	//Deallocate on device
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

//Kernel
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
#if (KERNEL_DEBUG ==1)
	printf("Hello, world from the device!\n");
#endif
	//One element of C computed by one thread in the kernel.
	specified_precision thread_c_value = 0;
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	for (int row_col_idx = 0; row_col_idx < A.width; row_col_idx++)
	{
		thread_c_value += A.elements[row*A.width + row_col_idx]*B.elements[row_col_idx*B.width + col];
	}
	C.elements[row*C.width+col] = thread_c_value;
}

//Notes:

//Each thread reads one row of A and one column of B, and in this implementation, A is read from global memory (B.width) times,
//while B is read from global memory (A.height) times. (This isn't actually entirely clear...).
