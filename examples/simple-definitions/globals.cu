__constant__ float constData[256];
float data[256];
hipMemcpyToSymbol(HIP_SYMBOL(constData), data, sizeof(data));
hipMemcpyFromSymbol(data, HIP_SYMBOL(constData), sizeof(data));

__device__ float devData;
float value = 3.14f;
hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float));

__device__ float * devPointer;
float* ptr;
hipMalloc(&ptr, 256*sizeof(float));
hipMemcpyToSymbol(HIP_SYMBOL(devPointer), &ptr, sizeof(ptr));
